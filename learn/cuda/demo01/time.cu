#include "hip/hip_runtime.h"
#include "cuda_common.cuh"
#include <stdio.h>

#include "/home/gxt_kt/Projects/debugstream/debugstream.hpp"

__global__ void hello_from_gpu() {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;

  const int id = threadIdx.x + blockIdx.x * blockDim.x;
  printf("Hello World from block %d and thread %d, global id %d\n", bid, tid,
         id);
}

int main(void) {

  CUDAERRORCHECK(hipSetDevice(1));

  // 使用event计算时间
  float time_elapsed = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start); // 创建Event
  hipEventCreate(&stop);

  hipEventRecord(start, 0); // 记录当前时间
  //===================
  // ....代码执行处
  hello_from_gpu<<<2, 4>>>();
  hipDeviceSynchronize();
  //===================
  hipEventRecord(stop, 0); // 记录当前时间

  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  hipEventDestroy(start);                          // destory the event
  hipEventDestroy(stop);

  printf("执行时间：%f(ms)\n", time_elapsed);

  return 0;
}