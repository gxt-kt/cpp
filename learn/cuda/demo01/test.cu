
#include <hip/hip_runtime.h>
#include <stdio.h>

// #include <iostream>

__global__ void hello_from_gpu() {
  // 核函数不支持iostream
  // std::cout << "hello cuda" << std::endl;
  printf("Hello World from the the GPU\n");
}

int main(void) {
  hello_from_gpu<<<4, 4>>>(); // 第一个指线程块数，第二个指每个线程块有多少线程
  {
    hipError_t cudaerr = hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    if (cudaerr != hipSuccess) {
      printf("kernel launch failed with error \"%s\".\n",
             hipGetErrorString(cudaerr));
    }
  }
  return 0;
}
