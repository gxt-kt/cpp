#include "hip/hip_runtime.h"
#include "/home/gxt_kt/Projects/debugstream/debugstream.hpp"
#include <stdio.h>

__global__ void Add(float *a, float *b, float *c, size_t cnts) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= cnts)
    return;
  *(c + tid) = *(a + tid) + *(b + tid);
}

int main() {
  int gpu_counts = 0;
  hipError_t err = hipGetDeviceCount(&gpu_counts);
  if (err != hipSuccess) {
    gDebug(hipGetErrorString(err));
  }
  gDebug(gpu_counts);
  hipSetDevice(gpu_counts);

  const int cnts = 1025;
  const int bytes = cnts * sizeof(float);

  auto host_a = std::shared_ptr<float[]>(new float[cnts]);
  auto host_b = std::shared_ptr<float[]>(new float[cnts]);
  auto host_c = std::shared_ptr<float[]>(new float[cnts]);
  memset(host_a.get(), 0, bytes);
  memset(host_b.get(), 0, bytes);
  memset(host_c.get(), 0, bytes);

  float *cuda_a;
  float *cuda_b;
  float *cuda_c;
  hipMalloc(&cuda_a, bytes);
  hipMalloc(&cuda_b, bytes);
  hipMalloc(&cuda_c, bytes);
  hipMemset(cuda_a, 0, bytes);
  hipMemset(cuda_b, 0, bytes);
  hipMemset(cuda_c, 0, bytes);

  for (int i = 0; i < cnts; i++) {
    host_a[i] = i * 0.1;
    host_b[i] = i * 0.2;
  }

  hipMemcpy(cuda_a, host_a.get(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(cuda_b, host_b.get(), bytes, hipMemcpyHostToDevice);

  dim3 grid(32, 1, 1);
  dim3 block((cnts + 32 - 1) / 32, 1, 1);
  Add<<<grid, block>>>(cuda_a, cuda_b, cuda_c, cnts);

  hipDeviceSynchronize();
  hipMemcpy(host_c.get(), cuda_c, bytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < cnts; i++) {
    gDebug() << VAR(i, host_c.get()[i]);
  }

  hipFree(cuda_a);
  hipFree(cuda_b);
  hipFree(cuda_c);

  hipDeviceReset();

  return 0;
}
