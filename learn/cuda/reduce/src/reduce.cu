#include "hip/hip_runtime.h"
#include "common.cuh"

__global__ void Reduce(int *data, int n) {
  // int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = GetTid();

#define REDUCE_TYPE 2

#if REDUCE_TYPE == 0
  if (tid == 0) {
    for (int i = 1; i < n; i++) {
      data[0] += data[i];
    }
  }
#elif REDUCE_TYPE == 1
  for (int stride = 1; stride < n; stride *= 2) {
    if (tid % (2 * stride) == 0) {
      data[tid] += data[tid + stride];
    }
    __syncthreads();
  }
#elif REDUCE_TYPE == 2
  for (int cnt = n / 2; cnt >= 1; cnt /= 2) {
    if (tid >= cnt) {
      return;
    }
    data[tid] += data[tid + cnt];
    __syncthreads();
  }
#endif
}

int main() {
  SetGPU();
  const int n = 8192;
  std::shared_ptr<int[]> nums = std::shared_ptr<int[]>(new int[n]);
  for (int i = 0; i < n; i++) {
    nums[i] = 1 + i;
  }
  int *cuda_nums;
  CUDAERRORCHECK(hipMalloc(&cuda_nums, n * sizeof(int)));
  CUDAERRORCHECK(hipMemcpy(cuda_nums, nums.get(), n * sizeof(int),
                            hipMemcpyHostToDevice));

  dim3 grid(n / 512);
  dim3 blocks(512);
  TIME_BEGIN_US(reduce);
  Reduce<<<grid, blocks>>>(cuda_nums, n);
  hipDeviceSynchronize();
  TIME_END(reduce);

  int result;
  CUDAERRORCHECK(
      hipMemcpy(&result, cuda_nums, 1 * sizeof(int), hipMemcpyDeviceToHost));
  gDebug(result);

  hipFree(cuda_nums);
  return 0;
}
